#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "ZoomOrRotate.cuh"

using byte = unsigned char;

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void Rotate_kernel(byte *GPU_source, byte *GPU_result, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight, double angle,
	int source_pitch, int source_pixelSize, int handle_pitch, int handle_pixelSize)
{
	//x��y���꼴Ϊ��Ӧ���߳�������thread�������x��y����
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= HandleWidth || y >= HandleHeight)
	{
		return;
	}

	//ӳ���ԭͼ�е�����
	point_double originCenter(SourceWidth / 2.0, SourceHeight / 2.0);
	point_double nowCenter(HandleWidth / 2.0, HandleHeight / 2.0);

	point now(x, y);
	point_double origin = getOriginProjection(now, nowCenter, originCenter, angle);

	if (origin.x >= SourceWidth || origin.y >= SourceHeight || origin.x < 0 || origin.y < 0)
	{
		return;
	}

	if (origin.x < 1 || origin.y < 1 || origin.x >= SourceWidth - 2 || origin.y >= SourceHeight - 2)
	{
		//ͼ���Եʹ������ڲ�ֵ��
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize] = GPU_source[(SourceHeight - 1 - (int)origin.y)*(-1)*source_pitch + (int)origin.x * source_pixelSize];
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 1] = GPU_source[(SourceHeight - 1 - (int)origin.y)*(-1)*source_pitch + (int)origin.x * source_pixelSize + 1];
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 2] = GPU_source[(SourceHeight - 1 - (int)origin.y)*(-1)*source_pitch + (int)origin.x * source_pixelSize + 2];
	}
	else
	{
		//����ٽ���
		point neighbors[4][4];
		getNeighborPoints(neighbors, origin.x, origin.y);

		//���Ȩֵ
		double weights[4][4];
		getWeight(weights, origin.x, origin.y);

		double value_R = 0;
		double value_G = 0;
		double value_B = 0;
		for (int y = 0; y < 4; y++)
		{
			for (int x = 0; x < 4; x++)
			{
				value_R += GPU_source[(SourceHeight - 1 - neighbors[x][y].y)*(-1)*source_pitch + neighbors[x][y].x * source_pixelSize] * weights[x][y];
				value_G += GPU_source[(SourceHeight - 1 - neighbors[x][y].y)*(-1)*source_pitch + neighbors[x][y].x * source_pixelSize + 1] * weights[x][y];
				value_B += GPU_source[(SourceHeight - 1 - neighbors[x][y].y)*(-1)*source_pitch + neighbors[x][y].x * source_pixelSize + 2] * weights[x][y];
			}
		}

		value_R = value_R < 0 ? 0 : value_R;
		value_R = value_R > 255 ? 255 : value_R;
		value_G = value_G < 0 ? 0 : value_G;
		value_G = value_G > 255 ? 255 : value_G;
		value_B = value_B < 0 ? 0 : value_B;
		value_B = value_B > 255 ? 255 : value_B;
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize] = (byte)value_R;
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 1] = (byte)value_G;
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 2] = (byte)value_B;
	}
}

extern "C" void Rotate_host(byte* source, byte* result_buf, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight, double angle,
	int source_pitch, int source_pixelSize, int handle_pitch, int handle_pixelSize)
{
	//ָ��GPU����ռ䷽ʽ
	dim3 DimBlock(BlockXMaxThreadNum, BlockYMaxThreadNum);
	dim3 DimGrid(HandleWidth / BlockXMaxThreadNum + 1, HandleHeight / BlockYMaxThreadNum + 1);

	//�������Դ��н��в�����ָ��
	byte* GPU_source;
	byte* GPU_result;

	//���Դ���Ϊԭͼ��͹���������ռ�
	checkCudaErrors(hipMalloc((void **)&GPU_source, sizeof(byte)*SourceHeight*((-1)*source_pitch)));
	checkCudaErrors(hipMalloc((void **)&GPU_result, sizeof(byte)*HandleHeight*((-1)*handle_pitch)));

	checkCudaErrors(hipMemcpy(GPU_source, source, sizeof(byte)*SourceHeight*((-1)*source_pitch), hipMemcpyHostToDevice));

	Rotate_kernel << < DimGrid, DimBlock >> > (GPU_source, GPU_result, HandleWidth, HandleHeight, SourceWidth, SourceHeight, angle,
		source_pitch, source_pixelSize, handle_pitch, handle_pixelSize);

	checkCudaErrors(hipMemcpy(result_buf, GPU_result, sizeof(byte)*HandleHeight*((-1)*handle_pitch), hipMemcpyDeviceToHost));

	hipFree(GPU_source);
	hipFree(GPU_result);
}

