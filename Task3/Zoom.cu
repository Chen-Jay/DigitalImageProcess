#include "hip/hip_runtime.h"
#pragma once

#include "ZoomOrRotate.cuh"

using byte = unsigned char;

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void Zoom_kernel(byte *GPU_source, byte *GPU_result, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight, double param)
{
	//x��y���꼴Ϊ��Ӧ���߳�������thread�������x��y����
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= HandleWidth || y >= HandleHeight)
	{
		return;
	}

	//ӳ���ԭͼ�е�����
	double x_origin = x / param;
	double y_origin = y / param;

	if (x_origin >= SourceWidth || y_origin >= SourceHeight)
	{
		return;
	}

	if (x_origin <= 1 || y_origin <= 1 || x_origin >= SourceWidth - 2 || y_origin >= SourceHeight - 2)
	{
		//ͼ���Եʹ������ڲ�ֵ��
		GPU_result[y*HandleWidth + x] = GPU_source[(int)y_origin*SourceWidth + (int)x_origin];
	}
	else
	{
		//����ٽ���
		point neighbors[4][4];
		getNeighborPoints(neighbors, x_origin, y_origin);
		//���Ȩֵ
		double weights[4][4];
		getWeight(weights, x_origin, y_origin);
		
		double value = 0;
		for (int y = 0; y < 4; y++)
		{
			for (int x = 0; x < 4; x++)
			{
				value += GPU_source[neighbors[x][y].y*SourceWidth + neighbors[x][y].x] * weights[x][y];
			}
		}

		value = value < 0 ? 0 : value;
		value = value > 255 ? 255 : value;
		GPU_result[y*HandleWidth + x] = (byte)value;
	}
}

extern "C" void Zoom_host(byte* source, byte* result_buf, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight)
{
	//�������ű�
	double param = (double)HandleHeight / SourceHeight;

	//ָ��GPU����ռ䷽ʽ
	dim3 DimBlock(BlockXMaxThreadNum, BlockYMaxThreadNum);
	dim3 DimGrid(HandleWidth / BlockXMaxThreadNum + 1, HandleHeight / BlockYMaxThreadNum + 1);

	//�������Դ��н��в�����ָ��
	byte* GPU_source;
	byte* GPU_result;

	//���Դ���Ϊԭͼ��͹���������ռ�
	checkCudaErrors(hipMalloc((void **)&GPU_source, sizeof(byte)*SourceWidth*SourceHeight));
	checkCudaErrors(hipMalloc((void **)&GPU_result, sizeof(byte)*HandleWidth*HandleHeight));

	checkCudaErrors(hipMemcpy(GPU_source, source, sizeof(byte)*SourceHeight*SourceWidth, hipMemcpyHostToDevice));

	Zoom_kernel << < DimGrid, DimBlock >> > (GPU_source, GPU_result, HandleWidth, HandleHeight, SourceWidth, SourceHeight, param);

	checkCudaErrors(hipMemcpy(result_buf, GPU_result, sizeof(byte)*HandleWidth*HandleHeight, hipMemcpyDeviceToHost));

	hipFree(GPU_source);
	hipFree(GPU_result);
}

