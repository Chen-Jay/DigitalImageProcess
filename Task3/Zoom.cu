#include "hip/hip_runtime.h"
#pragma once

#include "ZoomOrRotate.cuh"

using byte = unsigned char;

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void Zoom_kernel(byte *GPU_source, byte *GPU_result, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight, double param,
	int source_pitch, int source_pixelSize, int handle_pitch, int handle_pixelSize)
{
	//x��y���꼴Ϊ��Ӧ���߳�������thread�������x��y����
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= HandleWidth || y >= HandleHeight)
	{
		return;
	}

	//ӳ���ԭͼ�е�����
	double x_origin = x / param;
	double y_origin = y / param;

	if (x_origin >= SourceWidth || y_origin >= SourceHeight)
	{
		return;
	}

	if (x_origin <= 1 || y_origin <= 1 || x_origin >= SourceWidth - 2 || y_origin >= SourceHeight - 2)
	{
		//ͼ���Եʹ������ڲ�ֵ��
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize] = GPU_source[(SourceHeight - 1 - (int)y_origin)*(-1)*source_pitch + (int)x_origin * source_pixelSize];
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 1] = GPU_source[(SourceHeight - 1 - (int)y_origin)*(-1)*source_pitch + (int)x_origin * source_pixelSize + 1];
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 2] = GPU_source[(SourceHeight - 1 - (int)y_origin)*(-1)*source_pitch + (int)x_origin * source_pixelSize + 2];
	}
	else
	{
		//����ٽ���
		point neighbors[4][4];
		getNeighborPoints(neighbors, x_origin, y_origin);
		//���Ȩֵ
		double weights[4][4];
		getWeight(weights, x_origin, y_origin);

		double value_R = 0;
		double value_G = 0;
		double value_B = 0;

		for (int y = 0; y < 4; y++)
		{
			for (int x = 0; x < 4; x++)
			{
				value_R += GPU_source[(SourceHeight - 1 - neighbors[x][y].y)*(-1)*source_pitch + neighbors[x][y].x * source_pixelSize] * weights[x][y];
				value_G += GPU_source[(SourceHeight - 1 - neighbors[x][y].y)*(-1)*source_pitch + neighbors[x][y].x * source_pixelSize + 1] * weights[x][y];
				value_B += GPU_source[(SourceHeight - 1 - neighbors[x][y].y)*(-1)*source_pitch + neighbors[x][y].x * source_pixelSize + 2] * weights[x][y];
			}
		}

		value_R = value_R < 0 ? 0 : value_R;
		value_R = value_R > 255 ? 255 : value_R;
		value_G = value_G < 0 ? 0 : value_G;
		value_G = value_G > 255 ? 255 : value_G;
		value_B = value_B < 0 ? 0 : value_B;
		value_B = value_B > 255 ? 255 : value_B;
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize] = (byte)value_R;
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 1] = (byte)value_G;
		GPU_result[(HandleHeight - 1 - y)*(-1)*handle_pitch + x * handle_pixelSize + 2] = (byte)value_B;
	}
}

extern "C" void Zoom_host(byte* source, byte* result_buf, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight,
	int source_pitch, int source_pixelSize, int handle_pitch, int handle_pixelSize)
{
	//�������ű�
	double param = (double)HandleHeight / SourceHeight;

	//ָ��GPU����ռ䷽ʽ
	dim3 DimBlock(BlockXMaxThreadNum, BlockYMaxThreadNum);
	dim3 DimGrid(HandleWidth / BlockXMaxThreadNum + 1, HandleHeight / BlockYMaxThreadNum + 1);

	//�������Դ��н��в�����ָ��
	byte* GPU_source;
	byte* GPU_result;

	//���Դ���Ϊԭͼ��͹���������ռ�
	checkCudaErrors(hipMalloc((void **)&GPU_source, sizeof(byte)*SourceHeight*((-1)*source_pitch)));
	checkCudaErrors(hipMalloc((void **)&GPU_result, sizeof(byte)*HandleHeight*((-1)*handle_pitch)));

	checkCudaErrors(hipMemcpy(GPU_source, source, sizeof(byte)*SourceHeight*((-1)*source_pitch), hipMemcpyHostToDevice));

	Zoom_kernel <<< DimGrid, DimBlock >>> (GPU_source, GPU_result, HandleWidth, HandleHeight, SourceWidth, SourceHeight, param,
		source_pitch,source_pixelSize,handle_pitch,handle_pixelSize);

	checkCudaErrors(hipMemcpy(result_buf, GPU_result, sizeof(byte)*HandleHeight*((-1)*handle_pitch), hipMemcpyDeviceToHost));

	hipFree(GPU_source);
	hipFree(GPU_result);
}

