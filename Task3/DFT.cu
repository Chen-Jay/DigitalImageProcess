#include "hip/hip_runtime.h"
#include "DFT.cuh"

using byte = unsigned char;

inline void checkCudaErrors(hipError_t err, char* tag) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s. %s\n", hipGetErrorString(err), tag);
		return;
	}
}

__global__ void DFT_kernel(byte *GPU_source, byte *GPU_result, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight, int pitch, int pixelSize)
{
	//Ƶ������ϵ�µ�u��v���꼴Ϊ��Ӧ���߳�������thread�������x��y����
	int v = blockIdx.x*blockDim.x + threadIdx.x;
	int u = blockIdx.y*blockDim.y + threadIdx.y;

	if (v >= HandleWidth || u >= HandleHeight)
	{
		return;
	}

	ComplexNumber result;
	double realpart=0;
	double imaginepart =0;
	double greyValue;

	for (int x = 0; x < SourceHeight; x++)
	{
		for (int y = 0; y < SourceWidth; y++)
		{
			greyValue = (double)GPU_source[x*SourceWidth + y];
			if ((x + y) & 1)
				greyValue = -1.0*greyValue;

			double factor = (double)u*x / (double)SourceHeight + (double)v * y / (double)SourceWidth;

			double realpart_buf = cos(-2 * PI*(factor));
			double imaginepart_buf =sin(-2 * PI*(factor));

			realpart += realpart_buf * greyValue;
			imaginepart += imaginepart_buf * greyValue;
		}
	}
	double result_norm = 15 * log(std::sqrt(realpart*realpart+ imaginepart * imaginepart) + 1);

	result_norm = result_norm < 0.0 ? 0.0 : result_norm;
	result_norm = result_norm > 255.0 ? 255.0 : result_norm;

	GPU_result[(SourceHeight - 1 - u)*(-1)*pitch + v * pixelSize] = (byte)result_norm;
	GPU_result[(SourceHeight - 1 - u)*(-1)*pitch + v * pixelSize + 1] = (byte)result_norm;
	GPU_result[(SourceHeight - 1 - u)*(-1)*pitch + v * pixelSize + 2] = (byte)result_norm;

	//GPU_result[u*SourceWidth + v] = GPU_source[u*SourceWidth +v]; 
}

extern "C" void DFT_host(byte* source, byte* result_buf, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight, int pitch, int pixelSize)
{

	hipDeviceSetLimit(hipLimitPrintfFifoSize, (size_t)1024 * 1024 * 1024);

	//ָ��GPU����ռ䷽ʽ
	dim3 DimBlock(BlockXMaxThreadNum, BlockYMaxThreadNum);
	dim3 DimGrid(HandleWidth / BlockXMaxThreadNum + 1, HandleHeight / BlockYMaxThreadNum + 1);

	byte* result;

	//�������Դ��н��в�����ָ��
	byte* GPU_source;

	//���Դ���Ϊԭͼ��͹���������ռ�
	checkCudaErrors(hipMalloc((void **)&GPU_source, sizeof(byte)*SourceWidth*SourceHeight), "a");
	checkCudaErrors(hipMalloc((void **)&result, sizeof(byte)*HandleHeight*((-1)*pitch)), "b");

	checkCudaErrors(hipMemcpy(GPU_source, source, sizeof(byte)*SourceHeight*SourceWidth, hipMemcpyHostToDevice), "c");
	hipDeviceSynchronize();
	DFT_kernel <<< DimGrid, DimBlock >>> (GPU_source, result, HandleWidth, HandleHeight, SourceWidth, SourceHeight, pitch, pixelSize);
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(result_buf, result, sizeof(byte)*HandleHeight*((-1) * pitch), hipMemcpyDeviceToHost), "d");

	hipFree(GPU_source);
	hipFree(result);
}