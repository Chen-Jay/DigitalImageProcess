#include "hip/hip_runtime.h"
#include "DFT.cuh"

using byte = unsigned char;

inline void checkCudaErrors(hipError_t err) //cuda error handle function
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA Runtime API error:%s.\n", hipGetErrorString(err));
		return;
	}
}

__global__ void DFT_kernel(byte *GPU_source, byte *GPU_result, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight)
{
	//Ƶ������ϵ�µ�u��v���꼴Ϊ��Ӧ���߳�������thread�������x��y����
	int v = blockIdx.x*blockDim.x + threadIdx.x;
	int u = blockIdx.y*blockDim.y + threadIdx.y;

	if (v >= HandleWidth || u >= HandleHeight)
	{
		return;
	}

	ComplexNumber result;
	double greyValue;

	for (int x = 0; x < SourceHeight; x++)
	{
		for (int y = 0; y < SourceWidth; y++)
		{
			greyValue = GPU_source[x*SourceWidth + y];
			if ((x + y) & 1)
				greyValue = -1.0*greyValue;
			double factor = (double)u*x / (double)SourceHeight + (double)v * y / (double)SourceWidth;
			ComplexNumber buf(cos(-2 * PI*(factor)), sin(-2 * PI*(factor)));
			result = result + (buf)*greyValue;
		}
	}

	double result_norm = 15*log(result.getNorm()+1);

	result_norm= result_norm < 0.0 ? 0.0 : result_norm;
	result_norm= result_norm > 255.0 ? 255.0 : result_norm;

	GPU_result[u*SourceWidth + v] = (byte)result_norm;

	//GPU_result[u*SourceWidth + v] = GPU_source[u*SourceWidth +v];
}

extern "C" void DFT_host(byte* source, byte* result_buf, int HandleWidth, int HandleHeight, int SourceWidth, int SourceHeight)
{
	//ָ��GPU����ռ䷽ʽ
	dim3 DimBlock(BlockXMaxThreadNum, BlockYMaxThreadNum);
	dim3 DimGrid(HandleWidth / BlockXMaxThreadNum + 1, HandleHeight / BlockYMaxThreadNum + 1);

	//�������Դ��н��в�����ָ��
	byte* GPU_source;
	byte* GPU_result;

	//���Դ���Ϊԭͼ��͹���������ռ�
	checkCudaErrors(hipMalloc((void **)&GPU_source, sizeof(byte)*SourceWidth*SourceHeight));
	checkCudaErrors(hipMalloc((void **)&GPU_result, sizeof(byte)*HandleWidth*HandleHeight));

	checkCudaErrors(hipMemcpy(GPU_source, source, sizeof(byte)*SourceHeight*SourceWidth, hipMemcpyHostToDevice));

	DFT_kernel << < DimGrid, DimBlock >> > (GPU_source, GPU_result, HandleWidth, HandleHeight, SourceWidth, SourceHeight);

	checkCudaErrors(hipMemcpy(result_buf, GPU_result, sizeof(byte)*HandleWidth*HandleHeight, hipMemcpyDeviceToHost));

	hipFree(GPU_source);
	hipFree(GPU_result);
}